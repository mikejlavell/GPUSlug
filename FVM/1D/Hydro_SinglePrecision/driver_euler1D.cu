#include "hip/hip_runtime.h"
/*CUDA Implementation of the 1D Slug Code, for limited spatial reconstructions! 
Written by Steven Reeves, University of California, Santa Cruz
May 10th, 2017
*/

/*-----------------------Library Dependencies!----------------------*/
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <cmath>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <time.h>

/*-----------------------Header Dependencies!----------------------*/
#include "Slug_helper.cuh"

/*---------------Main Program, calls the routines!------*/
int main(){
	//Useful Debugging Features
 hipError_t cudaStatus;
 // Choose which GPU to run on, change this on a multi-GPU system.
 cudaStatus = hipSetDevice(0);
 if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipSetDevice failed!");
 }
 // Grid and Block size for CUDA Kernels
  dim3 block(512,1,1);
  dim3 grid(ceil(GRID_SIZE/block.x), 1 , 1);
  float t,dt;
  int nStep,ioCounter;

  t = 0.0f;

  nStep = 0;
  ioCounter = 0;
  
 /* -------------------- Grid Initialization ------------------------------*/

  /*-------------Host Variables ----------------------*/
  float *gr_xCoord; //Coordinate System
  float3* gr_U; // conservative vars
  float3* gr_V; // primitive vars

  /*-----------------------Device Variables ----------------------------------*/
  float3* d_gr_U; // conservative vars
  float3* d_gr_V; // primitive vars

  float3* d_gr_vL;   // left Riemann states
  float3* d_gr_vR;   // right Riemann states
  float3* d_gr_flux; // fluxes

 /*--------------------- Allocate -------------------------------------------*/
  //Grid Variables
 // grid_alloc(gr_xCoord ,gr_U, gr_V, //Host
 // d_gr_U, d_gr_V, d_gr_vL, d_gr_vR, d_gr_flux); //Device
 size_t num_bytes = GRID_SIZE*sizeof(float3);  
 size_t num_doub = GRID_SIZE*sizeof(float); 
 //Allocating Host Variables
 gr_xCoord = (float*)malloc(num_doub);
 gr_U = (float3*)malloc(num_bytes);   
 gr_V = (float3*)malloc(num_bytes);
 
 
 //Allocating Device Variables
 CudaSafeCall(hipMalloc((void**)&d_gr_U, num_bytes));
 CudaSafeCall(hipMalloc((void**)&d_gr_V, num_bytes));
 CudaSafeCall(hipMalloc((void**)&d_gr_vR, num_bytes));
 CudaSafeCall(hipMalloc((void**)&d_gr_vL, num_bytes));
 CudaSafeCall(hipMalloc((void**)&d_gr_flux, num_bytes));
 hipMemset(d_gr_U, 0.0f, num_bytes);
 hipMemset(d_gr_V, 0.0f, num_bytes);
 hipMemset(d_gr_vL, 0.0f, num_bytes);
 hipMemset(d_gr_vR, 0.0f, num_bytes);
 hipMemset(d_gr_flux, 0.0f, num_bytes);
 /*-------------------  Simulation Initialization  -----------------------------*/
  sim_init(gr_V, gr_U, gr_xCoord);
  //Write Initial Condition
  io_writeOutput(ioCounter, gr_xCoord, gr_V);
  ioCounter += 1;
	clock_t tStart = clock();
 //int click = 0;
/* =========================== Simulate =========================================*/
while (t < sim_tmax){
	 //calculate time step
	 dt = cfl(gr_V);
     if ( fabsf(t - sim_tmax) <= dt ){
        dt = fabsf(t - sim_tmax);
      }
	//std::cout<<"Delta t = " << dt <<" Pressure = " <<gr_V[5000].z <<std::endl;
      
     //Transfer to GPU
	 transfer_to_gpu(gr_V, gr_U, d_gr_V, d_gr_U); 
 /*------------------  Reconstruct and Update  --------------------------------*/
    //Launches Kernel to reconstruct cell interface values in d_gr_vL and vR respectively
     soln_reconstruct_PLM<<<grid,block>>>(dt, d_gr_V, d_gr_vL, d_gr_vR);
    CudaCheckError();
    hipDeviceSynchronize();
    // And gets the Numerical Flux. 
    soln_getFlux<<<grid,block>>>(dt, d_gr_vL, d_gr_vR, d_gr_flux);
    CudaCheckError();
    hipDeviceSynchronize();
    //  Updates the solution. 
     soln_update<<<grid,block>>>(d_gr_U, d_gr_V, d_gr_flux, dt);  
    	CudaCheckError();
    	hipDeviceSynchronize();//*/
    //Launches Kernel to update solution.
     
     //Transfer to CPU
     transfer_to_cpu(d_gr_V, d_gr_U, gr_V, gr_U);  
     //call BC on Primitive vars
     bc_apply(gr_V);
     //Call BC on Conservative vars
     bc_apply(gr_U);
    //update your time and step count
     t += dt;
     nStep += 1;
	if(dt < 0.0f) break;
	if(nStep > 1e10) break;
}

printf("Time taken: %fs\n", (float)(clock() - tStart) / (ncores*CLOCKS_PER_SEC));
/*------------------- Write End-Time Solution ------------------------------*/
  io_writeOutput(ioCounter, gr_xCoord, gr_V);

/*--------------------- Free the Variables ------------------------------------*/
   //Deallocating Host Variables
 free(gr_xCoord);
 free(gr_U);
 free(gr_V);
 
 //Deallocating Device Variables
 hipFree(d_gr_U);
 hipFree(d_gr_V);
 hipFree(d_gr_vR);
 hipFree(d_gr_vL);
 hipFree(d_gr_flux);
}
