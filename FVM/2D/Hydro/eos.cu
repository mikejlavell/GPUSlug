#include "hip/hip_runtime.h"
/* This header file contains the functions calculate the Equation of State*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "definition.h"


__host__ __device__ void eos_cell(const double dens, const double eint, double &pres)
{
       pres = fmax((sim_gamma-1.)*dens*eint,1e-6);
}

